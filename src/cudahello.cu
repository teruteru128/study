
#ifdef HAVE_CONFIG_H
#include "config.h"
#endif
#ifdef CMAKE_CUDA_COMPILER
#include <cuda_runtime.h>
#include <device_launch_parameters.h>
#endif

#include <hip/hip_runtime.h>
#include <libintl.h>
#include <locale.h>
#include <stdio.h>
#include <stdlib.h>
#define _(str) gettext(str)

int main(void)
{
#ifdef CMAKE_CUDA_COMPILER
    int device = 0;
    cudaError_t t = cudaSuccess;
    printf(_("Hello World!\n"));
    t = cudaGetDeviceCount(&device);
    if (!t)
    {
        printf("%d, %d\n", t, device);
    }
    else
    {
        printf("Error : %s \n%s\n", cudaGetErrorName(t),
               cudaGetErrorString(t));
    }
#endif
    return EXIT_SUCCESS;
}
