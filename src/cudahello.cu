
#if defined(__GNUC__)
#if __GNUC__ < 8
#include "config.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <libintl.h>
#include <locale.h>
#define _(str) gettext(str)

int main(void)
{
  int device = 0;
  hipError_t t = hipSuccess;
  printf(_("Hello World!\n"));
  t = hipGetDeviceCount(&device);
  if (!t)
  {
    printf("%d, %d\n", t, device);
  }
  else
  {
    printf("Error : %s \n%s\n", hipGetErrorName(t), hipGetErrorString(t));
  }
  return EXIT_SUCCESS;
}
#endif
#endif
