#include "hip/hip_runtime.h"

#include "config.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include "gettext.h"
#define _(str)   gettext(str)
#define N_(str)  gettext_noop(str)
#include <locale.h>

#define N 200
#define DEVICE 0

__global__ void sum_of_array(float *arr1, float *arr2, float *arr3, int size)
{
  // https://gist.github.com/onionmk2/854c333829f047a5e86cfab5a0ccae3a
  // x y zの3次元を1次元に直す。zは xとyの重みが、yはxの重みが、xは1の重みがあると考える。
  int block_id = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * (gridDim.x) + blockIdx.x;
  int threadId = block_id * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
  arr3[threadId] = arr1[threadId] + arr2[threadId];
}

void initialize_array(float *arr, size_t size)
{
  for (size_t i = 0; i < size; i++)
  {
    arr[i] = (float)rand();
  }
}

/* https://nonbiri-tereka.hatenablog.com/entry/2017/04/11/081601 */
int main(void)
{
  float *arr1, *arr2, *arr3;
  float *d_arr1 = NULL, *d_arr2 = NULL, *d_arr3 = NULL;
  size_t n_byte = N * sizeof(float);
  int device = 0;
  hipError_t error = hipSuccess;
  error = hipGetDeviceCount(&device);
  if (error)
  {
    printf(_("Error : %s\n%s\n"), hipGetErrorName(error), hipGetErrorString(error));
    return EXIT_FAILURE;
  }
  else
  {
    printf(ngettext("Success : one cuda device\n", "Success : %d cuda devices\n", device), device);
  }
  hipDeviceProp_t prop;
  for (int dev = 0; dev < device; dev++)
  {
    hipGetDeviceProperties(&prop, dev);
    printf("%s compute capability ", prop.name);
    printf("%d.%d\n", prop.major, prop.minor);
  }
  device = DEVICE;
  hipGetDeviceProperties(&prop, device);
  printf("Using %s\n\n", prop.name);
  hipSetDevice(device);

  arr1 = (float *)malloc(n_byte);
  arr2 = (float *)malloc(n_byte);
  arr3 = (float *)malloc(n_byte);

  initialize_array(arr1, n_byte);
  initialize_array(arr2, n_byte);
  initialize_array(arr3, n_byte);

  fputs("start hipMalloc\n", stderr);
  hipMalloc((void **)&d_arr1, n_byte);
  fputs("d_arr1\n", stderr);
  hipMalloc((void **)&d_arr2, n_byte);
  fputs("d_arr2\n", stderr);
  hipMalloc((void **)&d_arr3, n_byte);
  fputs("finish hipMalloc\n", stderr);

  fputs("start hipMemcpy\n", stderr);
  hipMemcpy(d_arr1, arr1, n_byte, hipMemcpyHostToDevice);
  hipMemcpy(d_arr2, arr2, n_byte, hipMemcpyHostToDevice);
  hipMemcpy(d_arr3, arr3, n_byte, hipMemcpyHostToDevice);
  fputs("finish hipMemcpy\n", stderr);

  fputs("start kernel function\n", stderr);
  sum_of_array<<<(N + 255) / 256, 256>>>(d_arr1, d_arr2, d_arr3, n_byte);
  fputs("finish kernel function\n", stderr);
  hipMemcpy(arr3, d_arr3, n_byte, hipMemcpyDeviceToHost);
  size_t i = 0;
  for (; i < 100; i++)
  {
    fprintf(stderr, "%f\n", arr3[i]);
  }
  return EXIT_SUCCESS;
}
