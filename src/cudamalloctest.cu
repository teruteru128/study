#include "hip/hip_runtime.h"

#include "config.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>

#define N 200

__global__
void sum_of_array(float *arr1, float *arr2, float *arr3, int size)
{
  // https://gist.github.com/onionmk2/854c333829f047a5e86cfab5a0ccae3a
  // x y zの3次元を1次元に直す。zは xとyの重みが、yはxの重みが、xは1の重みがあると考える。
  int block_id =    blockIdx.z * (gridDim.x * gridDim.y)
      + blockIdx.y * (gridDim.x)
      + blockIdx.x;
  int threadId = block_id * (blockDim.x * blockDim.y * blockDim.z)
    + (threadIdx.z * (blockDim.x * blockDim.y))
    + (threadIdx.y * blockDim.x)
    + threadIdx.x;
    arr3[threadId] = arr1[threadId] + arr2[threadId];
}

void initialize_array(float *arr, size_t size){
    for (size_t i = 0; i < size; i++){
        arr[i] = (float)rand();
    }
}

/* https://nonbiri-tereka.hatenablog.com/entry/2017/04/11/081601 */
int main(void)
{
  float *arr1, *arr2, *arr3;
  float *d_arr1 = NULL, *d_arr2 = NULL, *d_arr3 = NULL;
  size_t n_byte = N * sizeof(float);

  arr1 = (float *)malloc(n_byte);
  arr2 = (float *)malloc(n_byte);
  arr3 = (float *)malloc(n_byte);

  initialize_array(arr1, n_byte);
  initialize_array(arr2, n_byte);
  initialize_array(arr3, n_byte);

  fputs("start hipMalloc\n", stderr);
  hipMalloc((void**)&d_arr1, n_byte);
  fputs("d_arr1\n", stderr);
  hipMalloc((void**)&d_arr2, n_byte);
  fputs("d_arr2\n", stderr);
  hipMalloc((void**)&d_arr3, n_byte);
  fputs("finish hipMalloc\n", stderr);

  fputs("start hipMemcpy\n", stderr);
  hipMemcpy(d_arr1, arr1, n_byte, hipMemcpyHostToDevice);
  hipMemcpy(d_arr2, arr2, n_byte, hipMemcpyHostToDevice);
  hipMemcpy(d_arr3, arr3, n_byte, hipMemcpyHostToDevice);
  fputs("finish hipMemcpy\n", stderr);

  fputs("start kernel function\n", stderr);
  sum_of_array<<<(N+255)/256, 256>>>(d_arr1, d_arr2, d_arr3, n_byte);
  fputs("finish kernel function\n", stderr);
  hipMemcpy(arr3, d_arr3, n_byte, hipMemcpyDeviceToHost);
  size_t i = 0;
  for(; i < 100; i++){
    fprintf(stderr, "%f\n", arr3[i]);
  }
  return EXIT_SUCCESS;
}
